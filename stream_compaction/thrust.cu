#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            // Create device vectors from input and output data
            thrust::device_vector<int> d_in(idata, idata + n);
            thrust::device_vector<int> d_out(n);

            timer().startGpuTimer();
            // Perform exclusive scan using Thrust
            thrust::exclusive_scan(d_in.begin(), d_in.end(), d_out.begin());
            timer().endGpuTimer();

            // Copy result back to output array
            thrust::copy(d_out.begin(), d_out.end(), odata);
        }
    }
}
