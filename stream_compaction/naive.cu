#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__
        __global__ void naiveScanKernel(int n, int offset, int *odata, const int *idata){
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n){
                return;
            }

            if (index >= offset){
                odata[index] = idata[index - offset] + idata[index];
            }
            else{
                odata[index] = idata[index];
            }
        }

         __global__ void naiveScanFirstRound(int n, int *odata, const int *idata){
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n){
                return;
            }
            if (index == 0){
                odata[index] = 0;
            }
            else if (index == 1){
                odata[index] = idata[index - 1];
            }
            else{
                odata[index] = idata[index - 1] + idata[index - 2];
            }
         }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata){
            int block_size = 512;
            dim3 fullBlocksPerGrid((block_size + n - 1) / block_size);

            // TODO
            int d_round = ilog2ceil(n);
            int *dstFirst;
            int *dstSecond;
            hipMalloc((void **)&dstFirst, n * sizeof(int));
            hipMalloc((void **)&dstSecond, n * sizeof(int));

            hipMemcpy(dstFirst, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();
            naiveScanFirstRound<<<fullBlocksPerGrid, block_size>>>(n, dstSecond, dstFirst);
            std::swap(dstFirst, dstSecond);

            for (int d = 1; d < d_round; d++){
                int d_offset = 1 << d; // 2, 4, 8
                naiveScanKernel<<<fullBlocksPerGrid, block_size>>>(n, d_offset, dstSecond, dstFirst);
                std::swap(dstFirst, dstSecond);
            }
            timer().endGpuTimer();
            // setFirstAsZero<<<1, 1>>>(dstFirst);
            hipMemcpy(odata, dstFirst, n * sizeof(int), hipMemcpyDeviceToHost);
            hipFree(dstFirst);
            hipFree(dstSecond);
        }
    }
}
